#include "hip/hip_runtime.h"
#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <CUDACopy.hpp>
#include <cstdio>

const unsigned DIMX = 512;
const unsigned DIMY = 512;
const size_t   SIZE = DIMX*DIMY*4;

void kernel(unsigned char* dev_out);

int main(void)
{
    unsigned char *dev_out;

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other fg::* object to be created successfully
     */
    fg::Window wnd(DIMX, DIMY, "Fractal Demo");
    wnd.makeCurrent();
    /* create an font object and load necessary font
     * and later pass it on to window object so that
     * it can be used for rendering text */
    fg::Font fnt;
#ifdef OS_WIN
    fnt.loadSystemFont("Calibri");
#else
    fnt.loadSystemFont("Vera");
#endif
    wnd.setFont(&fnt);

    /* Create an image object which creates the necessary
     * textures and pixel buffer objects to hold the image
     * */
    fg::Image img(DIMX, DIMY, fg::FG_RGBA, fg::u8);
    /* copy your data into the pixel buffer object exposed by
     * fg::Image class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */
    CUDA_ERROR_CHECK(hipMalloc((void**)&dev_out, SIZE));
    kernel(dev_out);
    fg::copy(img, dev_out);

    do {
        wnd.draw(img);
    } while(!wnd.close());

    CUDA_ERROR_CHECK(hipFree(dev_out));
    return 0;
}

__device__
int julia(int x, int y)
{

    const float scale = 1.5;
    float jx = scale * (float)(DIMX/2.0f - x)/(DIMX/2.0f);
    float jy = scale * (float)(DIMY/2.0f - y)/(DIMY/2.0f);

    hipFloatComplex c = make_hipFloatComplex(-0.8f, 0.156f);
    hipFloatComplex a = make_hipFloatComplex(jx, jy);

    for (int i=0; i<200; i++) {
        a = hipCaddf(hipCmulf(a, a), c);
        if (hipCabsf(a) > 1000.0f)
            return 0;
    }

    return 1;
}

__global__
void julia(unsigned char* out)
{
    int x = blockIdx.x * blockDim.x  + threadIdx.x;
    int y = blockIdx.y * blockDim.y  + threadIdx.y;

    if (x<DIMX && y<DIMY) {
        int offset = x + y * DIMX;
        // now calculate the value at that position
        int juliaValue = julia(x, y);

        out[offset*4 + 2] = 255 * juliaValue;
        out[offset*4 + 0] = 0;
        out[offset*4 + 1] = 0;
        out[offset*4 + 3] = 255;
    }
}

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

void kernel(unsigned char* dev_out)
{
    static const dim3 threads(8, 8);
    dim3 blocks(divup(DIMX, threads.x),
                divup(DIMY, threads.y));

    julia<<< blocks, threads >>>(dev_out);
}
