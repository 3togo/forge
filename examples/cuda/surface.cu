#include "hip/hip_runtime.h"
#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <CUDACopy.hpp>
#include <cstdio>
#include <iostream>

const float XMIN = -8.0f;
const float XMAX = 8.f;
const float YMIN = -8.0f;
const float YMAX = 8.f;

const float DX = 0.5;
const size_t XSIZE = (XMAX-XMIN)/DX;
const size_t YSIZE = (YMAX-YMIN)/DX;

void kernel(float dx, float* dev_out);

int main(void)
{
    float *dev_out;

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other fg::* object to be created successfully
     */
    fg::Window wnd(1024, 768, "3d Surface Demo");
    wnd.makeCurrent();

    fg::Chart chart(FG_CHART_3D);
    chart.setAxesLimits(-10.f, 10.f, -10.f, 10.f, -0.5f, 1.f);
    chart.setAxesTitles("x-axis", "y-axis", "z-axis");

    fg::Surface surf = chart.surface(XSIZE, YSIZE, f32);
    surf.setColor(FG_YELLOW);

    FORGE_CUDA_CHECK(hipMalloc((void**)&dev_out, XSIZE * YSIZE * 3 * sizeof(float) ));
    kernel(DX, dev_out);
    /* copy your data into the vertex buffer object exposed by
     * fg::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */
    fg::copy(surf.vertices(), dev_out);

    do {
        wnd.draw(chart);
    } while(!wnd.close());

    FORGE_CUDA_CHECK(hipFree(dev_out));
    return 0;
}


__global__
void sincos_surf(float dx, float* out,
				 const float XMIN, const float YMIN,
				 const size_t XSIZE, const size_t YSIZE)
{
    int i = blockIdx.x * blockDim.x  + threadIdx.x;
    int j = blockIdx.y * blockDim.y  + threadIdx.y;

    float x= ::XMIN + i*dx;
    float y= ::YMIN + j*dx;
    if (i<XSIZE && j<YSIZE) {
        int offset = j + i * YSIZE;
        out[ 3 * offset     ] = x;
        out[ 3 * offset + 1 ] = y;
        float z = sqrt(x*x+y*y) + 2.2204e-16;
        out[ 3 * offset + 2 ] = sinf(z)/z;
    }
}

inline
int divup(int a, int b)
{
    return (a+b-1)/b;
}

void kernel(float dx, float* dev_out)
{
    static const dim3 threads(8, 8);
    dim3 blocks(divup(XSIZE, threads.x),
                divup(YSIZE, threads.y));

    sincos_surf<<< blocks, threads >>>(dx, dev_out, XMIN, YMIN, XSIZE, YSIZE);
}
