#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <CUDACopy.hpp>
#include <cstdio>
#include <iostream>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;

static const float ZMIN = 0.1f;
static const float ZMAX = 10.f;

const float DX = 0.005;
const size_t ZSIZE = (ZMAX-ZMIN)/DX+1;

void kernel(float t, float dx, float* dev_out);

int main(void)
{
    float *dev_out;

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other fg::* object to be created successfully
     */
    fg::Window wnd(DIMX, DIMY, "Plot 3d Demo");
    wnd.makeCurrent();
    /* create an font object and load necessary font
     * and later pass it on to window object so that
     * it can be used for rendering text */
    fg::Font fnt;
#ifdef OS_WIN
    fnt.loadSystemFont("Calibri", 32);
#else
    fnt.loadSystemFont("Vera", 32);
#endif
    wnd.setFont(&fnt);

    /* Create several plot objects which creates the necessary
     * vertex buffer objects to hold the different plot types
     */
    fg::Plot3 plot3(ZSIZE, fg::f32);

    /*
     * Set draw limits for plots
     */
    plot3.setAxesLimits(1.1f, -1.1f, 1.1f, -1.1f, 10.f, 0.f);

    /*
    * Set axis titles
    */
    plot3.setAxesTitles("x-axis", "y-axis", "z-axis");

    static float t=0;
    CUDA_ERROR_CHECK(hipMalloc((void**)&dev_out, ZSIZE * 3 * sizeof(float) ));
    kernel(t, DX, dev_out);
    /* copy your data into the vertex buffer object exposed by
     * fg::Plot class and then proceed to rendering.
     * To help the users with copying the data from compute
     * memory to display memory, Forge provides copy headers
     * along with the library to help with this task
     */
    fg::copy(plot3, dev_out);


    do {
        t+=0.01;
        kernel(t, DX, dev_out);
        fg::copy(plot3, dev_out);
        // draw window and poll for events last
        wnd.draw(plot3);
    } while(!wnd.close());

    CUDA_ERROR_CHECK(hipFree(dev_out));
    return 0;
}


__global__
void gen_curve(float t, float dx, float* out)
{
    int offset = blockIdx.x * blockDim.x  + threadIdx.x;

    float z = ZMIN + offset*dx;
    if(offset < ZSIZE){
        out[ 3 * offset     ] = cos(z*t+t)/z;
        out[ 3 * offset + 1 ] = sin(z*t+t)/z;
        out[ 3 * offset + 2 ] = z + 0.1*sin(t);
    }
}

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

void kernel(float t, float dx, float* dev_out)
{
    static const dim3 threads(1024);
    dim3 blocks(divup(ZSIZE, 1024));

    gen_curve<<< blocks, threads >>>(t, dx, dev_out);
}
