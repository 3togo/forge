#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2015-2019, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <forge.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <CUDACopy.hpp>
#include <cstdio>

const unsigned DIMX = 1000;
const unsigned DIMY = 800;
const unsigned IMG_SIZE = DIMX * DIMY * 4;
const unsigned WIN_ROWS = 1;
const unsigned WIN_COLS = 2;

const unsigned NBINS = 5;


const static int hperm[] = {26, 58, 229, 82, 132, 72, 144, 251, 196, 192, 127, 16,
    68, 118, 104, 213, 91, 105, 203, 61, 59, 93, 136, 249, 27, 137, 141, 223, 119,
    193, 155, 43, 71, 244, 170, 115, 201, 150, 165, 78, 208, 53, 90, 232, 209, 83,
    45, 174, 140, 178, 220, 184, 70, 6, 202, 17, 128, 212, 117, 200, 254, 57, 248,
    62, 164, 172, 19, 177, 241, 103, 48, 38, 210, 129, 23, 211, 8, 112, 107,  126,
    252,  198, 32, 123, 111,  176,  206, 15, 219, 221, 147, 245, 67, 92, 108, 143,
    54, 102, 169, 22, 74, 124, 181, 186, 138, 18, 7, 34, 81, 46, 120, 236, 89,228,
    197, 205, 13, 63, 134,  242, 157, 135, 237, 35, 234, 49, 85, 76, 148, 188, 98,
    87, 173, 84, 226, 11, 125, 122, 2, 94, 191, 179, 175, 187, 133, 231, 154,  44,
    28, 110, 247, 121, 146, 240, 97, 88, 130,195, 30, 25, 56, 171, 80, 69, 139, 9,
    238, 160, 227, 204, 31, 40, 66, 77, 21, 159,  162, 207,  167, 214, 10, 3, 149,
    194, 239, 166,  145, 235, 20, 50, 113, 189, 99, 37, 86, 42, 168, 114, 96, 246,
    183, 250, 233, 156, 52,  65, 131, 47,  255, 5, 33, 217, 73, 4, 60, 64, 109, 0,
    215, 100, 180, 12, 24, 190, 222, 106, 41, 216, 230, 161, 55, 152, 79, 75, 142,
    36, 101, 1, 253, 225, 51, 224, 182, 116, 218, 95, 39, 158,  14, 243, 151, 163,
    29, 153, 199, 185
};

__constant__ int perm[256];
void kernel(unsigned char* dev_out);
void kernel_hist(unsigned char * src, int* hist_out);

int main(void)
{
    int *hist_out;
    unsigned char *dev_out;
    hipMemcpyToSymbol(HIP_SYMBOL(perm), hperm, 256 * sizeof(int));

    /*
     * First Forge call should be a window creation call
     * so that necessary OpenGL context is created for any
     * other fg::* object to be created successfully
     */
    fg::Window wnd(DIMX, DIMY, "Histogram Demo");
    wnd.makeCurrent();
    /* create an font object and load necessary font
     * and later pass it on to window object so that
     * it can be used for rendering text */
    fg::Font fnt;
#ifdef OS_WIN
    fnt.loadSystemFont("Calibri", 32);
#else
    fnt.loadSystemFont("Vera", 32);
#endif
    wnd.setFont(&fnt);

    /*
     * Split the window into grid regions
     */
    wnd.grid(WIN_ROWS, WIN_COLS);

    fg::Image img(DIMX, DIMY, fg::FG_RGBA, fg::FG_UNSIGNED_BYTE);
    /*
     * Create histogram object while specifying desired number of bins
     */
    fg::Histogram hist(NBINS, fg::FG_UNSIGNED_BYTE);

    /*
     * Set histogram colors
     */
    hist.setBarColor(fg::FG_YELLOW);

    /* set x axis limits to maximum and minimum values of data
     * and y axis limits to range [0, nBins]*/
    hist.setAxesLimits(1, 0, 1000, 0);
    CUDA_ERROR_CHECK(hipMalloc((void**)&dev_out, IMG_SIZE ));
    CUDA_ERROR_CHECK(hipMalloc((void**)&hist_out, NBINS));
    kernel(dev_out);
    kernel_hist(dev_out, hist_out);
    fg::copy(img, dev_out);
    fg::copy(hist, hist_out);

    do {
        kernel(dev_out);
        kernel_hist(dev_out, hist_out);
        fg::copy(img, dev_out);
        fg::copy(hist, hist_out);
        wnd.draw(0, 0, img,  NULL );
        wnd.draw(1, 0, hist, NULL );
        wnd.draw();
    } while(!wnd.close());

    CUDA_ERROR_CHECK(hipFree(dev_out));
    return 0;
}

__device__
inline float interp(float t){
    return ((6 * t - 15) * t + 10) * t * t * t;
}

__device__
inline float lerp (float x0, float x1, float t) {
        return x0 + (x1 - x0) * t;
}

__device__
inline float dot (float2 v0, float2 v1) {
        return v0.x*v1.x + v0.y*v1.y;
}

__device__
inline float2 sub (float2 v0, float2 v1) {
        return make_float2(v0.x-v1.x, v0.y-v1.y);
}

__device__
float perlinNoise(float x, float y, int tileSize) {
    const float2 default_gradients[] = { make_float2(1,1), make_float2(-1,1),make_float2 (1,-1), make_float2(-1,-1) };
    int x_grid = x/tileSize;
    int y_grid = y/tileSize;
    unsigned rand_id0 = perm[(x_grid+2*y_grid) % 256 ] % 4;
    unsigned rand_id1 = perm[(x_grid+1+2*y_grid) % 256 ] % 4;
    unsigned rand_id2 = perm[(x_grid+2*(y_grid+1)) % 256 ] % 4;
    unsigned rand_id3 = perm[(x_grid+1+2*(y_grid+1)) % 256 ] % 4;

    x=fmod(x,__int2float_rd(tileSize))/tileSize;
    y=fmod(y,__int2float_rd(tileSize))/tileSize;
    float u = interp(x);
    float v = interp(y);

    float influence_vecs[4];
    influence_vecs[0] = dot(sub(make_float2(x,y), make_float2(0,0)), default_gradients[rand_id0]);
    influence_vecs[1] = dot(sub(make_float2(x,y), make_float2(1,0)), default_gradients[rand_id1]);
    influence_vecs[2] = dot(sub(make_float2(x,y), make_float2(0,1)), default_gradients[rand_id2]);
    influence_vecs[3] = dot(sub(make_float2(x,y), make_float2(1,1)),             default_gradients[rand_id3]);

    return lerp(lerp(influence_vecs[0], influence_vecs[1], u), lerp(influence_vecs[2], influence_vecs[3], u), v);

}
 __device__
float octavesPerlin(float x, float y, int octaves, float persistence, int tileSize) {
    float total = 0, max_value = 0;
    float amplitude = 1, frequency = 1;
    for(int i=0; i<octaves; ++i){
        total += perlinNoise( x*frequency, y*frequency, tileSize) * amplitude;
        max_value += amplitude;
        amplitude *= persistence;
        frequency *= 2;
    }
    return total/max_value;
}

inline int divup(int a, int b)
{
    return (a+b-1)/b;
}

__global__
void image_gen(unsigned char* out, float persistance, int tileSize) {
    int x = blockIdx.x * blockDim.x  + threadIdx.x;
    int y = blockIdx.y * blockDim.y  + threadIdx.y;

    if (x<DIMX && y<DIMY) {
        int offset  = y * DIMX + x;
        int octaves = 4;
        int noiseValue    = 255 * octavesPerlin(x, y, octaves, persistance, tileSize);
        out[offset*4 + 0] = noiseValue;
        out[offset*4 + 1] = noiseValue;
        out[offset*4 + 2] = noiseValue;
        out[offset*4 + 3] = 255;
    }
}

void kernel(unsigned char* dev_out) {
    static const dim3 threads(8, 8);
    dim3 blocks(divup(DIMX, threads.x),
                divup(DIMY, threads.y));

    static int tileSize = 32; tileSize++;
    static float persistance = 0.1; persistance+=0.01;
    image_gen<<< blocks, threads >>>(dev_out, persistance, tileSize);
}

__global__
void hist_freq(const unsigned char* src, int* hist_array, const unsigned nbins) {
    int x = blockIdx.x * blockDim.x  + threadIdx.x;
    int y = blockIdx.y * blockDim.y  + threadIdx.y;

    if (x<DIMX && y<DIMY) {
        int offset  = y * DIMX + x;
        unsigned char noiseVal = src[offset*4 + 0];
        atomicAdd(hist_array + __float2int_rd(nbins * __int2float_rd(noiseVal)/255.f), 1);
    }
}

void kernel_hist(unsigned char * src, int* hist_out){
    static const dim3 threads(NBINS);
    dim3 blocks(1);
    hipMemset(hist_out, 0, NBINS * sizeof(int));
    hist_freq<<< blocks, threads >>>(src, hist_out, NBINS);
}
